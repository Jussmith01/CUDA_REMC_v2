#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <sstream>
#include <string>
#include <string.h>
#include <fstream>
#include "lib_cuda_main.h"
#include "lib_mpi.h"
#include "lib_classes.h"
#include "lib_cuda_functions.h"

using namespace std;

//___________________________________________________________________//
// ----------------------------------------------------------------- //
// ------------------MC EVOLUTION STEP(EACH REPLICA)---------------- //
// ----------------------------------------------------------------- //
//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++//
extern void build_iState_array_cuda(int *Sa,double *Ta,double *Ea,double Tmax,double Tmin,int Tinc,int N,float *gpu_t)
{
        int *d_Sa,*Ra,*d_Ra;
        unsigned int *ModB;
        double *Tincsize,*Tl,*d_Ta,*d_Ea,*EaT,*d_EaT;
        double *d_Tincsize,*d_Tmin;

        //hipSetDevice(1);

        //******DEFINE THREADS AND BLOCK SIZES*******
	int blocks, threads;
	Get_threads_blocks(threads,blocks,Tinc,N);

        //*****MEMORY ALLOCATIONS CPU******
        Tincsize = (double *)malloc(sizeof(double));
        Tl = (double *)malloc(sizeof(double));
        int RaSize = N * N * Tinc; Ra = (int *)malloc(RaSize * sizeof(int));
        ModB = (unsigned int *)malloc(sizeof(unsigned int)); ModB[0] = (unsigned int)N;
        EaT = (double *)malloc(N * N * Tinc * sizeof(double));

        //*****MEMORY ALLOCATIONS GPU*******    
        hipError_t (hipMalloc ((void **) &d_Tincsize, sizeof(double)));
        hipError_t (hipMalloc ((void **) &d_Sa, Tinc * N * N * sizeof(int)));
        hipError_t (hipMalloc ((void **) &d_Ra, Tinc * N * N * sizeof(int)));
        hipError_t (hipMalloc ((void **) &d_Ta, Tinc * sizeof(double)));
        hipError_t (hipMalloc ((void **) &d_Ea, Tinc * sizeof(double)));
        hipError_t (hipMalloc ((void **) &d_Tmin, sizeof(double)));
        hipError_t (hipMalloc ((void **) &d_EaT, N * N  * Tinc * sizeof(double)));

        hipMemcpyToSymbol(HIP_SYMBOL(d_ModB), ModB, sizeof(unsigned int));
        hipError_t codeMemA1 = hipGetLastError();
        if (codeMemA1 != hipSuccess)
                printf("Cuda error INITIAL MEM ALLOCATION COPY -- %s\n",hipGetErrorString(codeMemA1));

        //********DEFINE NEEDED VALUES*******
	//double a = 1;
        //Tincsize[0] = (log(Tmax)/a - log(Tmin)/a) / Tinc;
	Tincsize[0] = (Tmax - Tmin) / Tinc;
        //Tl[0] = log(Tmin)/a;
	Tl[0] = Tmin;
        random_ints(Ra,RaSize,2,1);

        //*****Create events for GPU timer*****
        hipEvent_t event1, event2;
        hipError_t(hipEventCreate(&event1));
        hipError_t(hipEventCreate(&event2));

        /* ... Load CPU data into GPU buffers  */
        hipError_t (hipMemcpy(d_Tincsize, Tincsize, sizeof(double), hipMemcpyHostToDevice));
        hipError_t (hipMemcpy(d_Tmin, Tl, sizeof(double), hipMemcpyHostToDevice));
        hipError_t (hipMemcpy(d_Ra, Ra,RaSize * sizeof(int), hipMemcpyHostToDevice));
        hipError_t (hipMemcpy(d_Ea,Ea,Tinc * sizeof(double),hipMemcpyHostToDevice));
        hipError_t (hipMemcpy(d_EaT,EaT, N * N * Tinc * sizeof(double),hipMemcpyHostToDevice));
        hipError_t codeMem1 = hipGetLastError();
        if (codeMem1 != hipSuccess)
                printf("Cuda error INITIAL MEM COPY -- %s\n",hipGetErrorString(codeMem1));

        //*****RUN PROGRAMS(EVENTS TIME GPU CALCULATIONS)****
        hipEventRecord(event1,0);

        __temper_array_cuda__ <<<Tinc,1>>> (d_Ta,d_Tmin,d_Tincsize);
        hipDeviceSynchronize();
        hipError_t code = hipGetLastError();
        if (code != hipSuccess)
                printf("Cuda BUILD INITIAL TEMPER error -- %s\n",hipGetErrorString(code));

        __iState_array_cuda__ <<<blocks,threads>>> (d_Sa,d_Ra);
        hipDeviceSynchronize();
        hipError_t code2 = hipGetLastError();
        if (code2 != hipSuccess)
                printf("Cuda BUILD ISTATE  error -- %s\n",hipGetErrorString(code2));

        __CalcE_array_cuda__ <<<blocks,threads>>> (d_Sa,d_EaT);
        hipDeviceSynchronize();
        hipError_t code3 = hipGetLastError();
        if (code3 != hipSuccess)
                printf("Cuda INITIAL E CALC error -- %s\n",hipGetErrorString(code3));

        __sumE_array_cuda__ <<<1,Tinc>>> (d_Ea,d_EaT);
        hipDeviceSynchronize();
        hipError_t code4 = hipGetLastError();
        if (code != hipSuccess)
                printf("Cuda SUM E ERROR  error -- %s\n",hipGetErrorString(code4));

        hipError_t (hipEventRecord(event2,0));

        /* ... Transfer data from GPU to CPU */
        hipError_t (hipMemcpy(Sa,d_Sa,Tinc * N * N * sizeof(int),hipMemcpyDeviceToHost));
        hipError_t (hipMemcpy(Ta,d_Ta,Tinc * sizeof(double),hipMemcpyDeviceToHost));
        hipError_t (hipMemcpy(Ea,d_Ea,Tinc * sizeof(double),hipMemcpyDeviceToHost));

        hipError_t (hipMemcpy(Tincsize, d_Tincsize, sizeof(double), hipMemcpyDeviceToHost));
        hipError_t (hipMemcpy(Tl, d_Tmin, sizeof(double), hipMemcpyDeviceToHost));
        hipError_t (hipMemcpy(Ra, d_Ra,RaSize * sizeof(int), hipMemcpyDeviceToHost));
        hipError_t (hipMemcpy(EaT,d_EaT, N * N * Tinc * sizeof(double),hipMemcpyDeviceToHost));

	
        hipError_t codeMemcp1 = hipGetLastError();
        if (codeMemcp1 != hipSuccess)
                printf("Cuda MEMORY 1 error -- %s\n",hipGetErrorString(codeMemcp1));
	
        //Sync the events so that 2 doesnt finish before the end of computations.
        hipError_t (hipEventSynchronize(event1));
        hipError_t (hipEventSynchronize(event2));

        //char* hipGetErrorString(hipError_t error);
        float dt_ms;
        hipError_t (hipEventElapsedTime(&dt_ms, event1, event2));
        hipError_t codetime = hipGetLastError();
        if (codetime != hipSuccess)
                printf("Cuda TIMER 1 error -- %s\n",hipGetErrorString(codetime));
	for (int i = 0; i < Tinc; ++i)
	{
		cout << "TEMP(" << i << ")= " << Ta[i] << "\n";
	}

        gpu_t[0] += (float) dt_ms;
        //************FREE MEMORY***********
        free(Tincsize); free(Tl); free(Ra); free(ModB); free(EaT);
        hipFree(d_Tincsize); hipFree(d_Sa); hipFree(d_Ta); hipFree(d_Tmin); hipFree(d_Ra); hipFree(d_Ea); hipFree(d_EaT);
}

//___________________________________________________________________//
// ----------------------------------------------------------------- //
// ------------------MC EVOLUTION STEP(EACH REPLICA)---------------- //
// ----------------------------------------------------------------- //
//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++//
extern void MCEvo_States_cuda(int *Sa,double *Ta,double *Ea,int N,int Tinc,ofstream &file1,int *REMC_parm,double convc,float *gpu_t,char *dir)
{
        //NOTE THAT REMC_parm contains the parameters for the run:
	//REMC_parm[i]: i = 0 -> MC cycles per RE, i = 1 -> Replica exchanges, i = 2 ->energy sample every x MC cycles
	
	int i,j,k,*perm_func,*Reps_Ex,pId_loc,*d_Sa;
	double *d_Ea,*d_EaT;
	struct  timespec timerN,timer0;
	histo_data *hist_dat;
	
	//Define run parameters -- #MCcyclesTotal = MCc*ESs*REs 
	int MCc = REMC_parm[0]; //Define MC cycles per RE
	int REs = REMC_parm[1]; //Define number of replica exchanges
	int ESs = REMC_parm[2]; //Define number of energy samples per cycle 
	int RepSw = REMC_parm[3]; //Replica exchange switch 0 turns it on 1 turns it off
	//Set time
	clock_gettime(CLOCK_REALTIME,&timer0);

	cout << "!!!!!REMC BEGINS!!!!!\n";
	//*************************************************************//
        //*******************MEMORY ALLOCATIONS (HOST)*****************//
	//*************************************************************//

        Reps_Ex = (int *)malloc(Tinc * sizeof(int));
	hist_dat = (histo_data *)malloc(Tinc * sizeof(histo_data));
        perm_func = (int *)malloc(Tinc * sizeof(int));

        //*****MEMORY ALLOCATIONS GPU*******    
        hipError_t(hipMalloc ((void **) &d_Sa, Tinc * N * N * sizeof(int)));
        hipError_t(hipMalloc ((void **) &d_Ea, Tinc * sizeof(double)));
        hipError_t(hipMalloc ((void **) &d_EaT, Tinc * N * N * sizeof(double)));
        hipError_t codeMemAlloc = hipGetLastError();
        if (codeMemAlloc != hipSuccess)
                printf("Cuda error ALLOCATIONS -- %s\n",hipGetErrorString(codeMemAlloc));


        //******PRINT CUDA DEVICE PROPERTIES*********//
 	Print_CUDA_Device_Props (file1);
	
        //******DEFINE THREADS AND BLOCK SIZES*******//
        int blocks, threads;
	Get_threads_blocks(threads,blocks,Tinc,N);
	file1 << "\nSetting CUDA block/thread count:\n";
        file1 << "Blocks: " << blocks << " Threads: " << threads  << "\n\n";
 	
	//*****Create events for GPU timer*****
	float dt_ms;
        hipEvent_t event3, event4;
        hipEventCreate(&event3);
        hipEventCreate(&event4);

	//SETUP CLASSES OBJECTS FOR HISTOGRAM DATA LOGGING AND OUTPUT FILES
	file1 << "Replica Temperatures:\n";
	for (i = 0; i < Tinc;++i)
	{
		hist_dat[i].reset_probs();
 		hist_dat[i].set_RE_prob();
		hist_dat[i].Temp = Ta[i];
		hist_dat[i].array_size = (REs * ESs);
        	hist_dat[i].set_Ealloc();
		perm_func[i] = i;
		Reps_Ex[i] = 0;
                file1 << "Temp("  << i << ")= " << hist_dat[i].Temp << "\n";
	}
	
//----------------------------------------------------------------------------//
        //*************************************************************//
        //*******************BEGIN REPLICA EXCHANGE MC*****************//
        //*************************************************************//
	file1 << "\n|------------------REMC STARTING------------------|\n";
        for (j = 0; j < REs; ++j)
        {
                cout << "|******BEGIN CYCLE " << j << "********|" << "\n";
		file1 << "|************Cycle " << j  << "************|" << "\n";	
                //*************RUN REPLICA EXCHANGE CODE*************
		switch(RepSw)
		{
			case 0:
			switch(j)
			{
				case 0:
				cout << "replica exchange skipped" << "\n";
				break;
			
				default:
                		Replic_Ex_omp(Ea,Ta,Reps_Ex,Tinc,hist_dat,perm_func,j,N,file1);
              			break;
			}
			break;

			default:
			file1 << "Replica Exchange Disabled\n";
			break;
                }
		//*****RUN MC PROGRAMS(EVENTS TIME GPU CALCULATIONS)****
			for (k = 0; k < ESs ; ++k)
			{ 
                        	CalcE_MCEvo_OMP(Sa, Ta, Tinc, MCc, N, hist_dat, perm_func, j);

				//***********SAMPLE THE ENERGY STATES**************
				hipEventRecord(event3,0);
				
				//-------Load states into the GPU buffers---------//
				hipError_t(hipMemcpy(d_Sa,Sa,Tinc * N * N * sizeof(int),hipMemcpyHostToDevice));				
				hipError_t codeMem = hipGetLastError();
                                if (codeMem != hipSuccess)
                                        printf("Cuda error Mem -- %s\n",hipGetErrorString(codeMem));
				
				//------CALCULATE THE ENERGY FOR HISTO DATA-------//
			        __CalcE_array_cuda__ <<<blocks,threads>>> (d_Sa,d_EaT);
        			hipDeviceSynchronize();
        			hipError_t code = hipGetLastError();
        			if (code != hipSuccess)
                			printf("Cuda error -- %s\n",hipGetErrorString(code));

			        __sumE_array_cuda__ <<<Tinc,1>>> (d_Ea,d_EaT);
        			hipDeviceSynchronize();
        			hipError_t code1 = hipGetLastError();
        			if (code1 != hipSuccess)
                		printf("Cuda error -- %s\n",hipGetErrorString(code1));
 				
				//--------END ENERGY CALC FOR HIST DATA-----------//	
			 	
				//------Load data from GPU to HOST------//
				 hipError_t(hipMemcpy(Ea,d_Ea,Tinc * sizeof(double),hipMemcpyDeviceToHost));				
				
				//-----------SAVE GPU TIMER DATA------------//
				hipEventRecord(event4,0);
				hipEventSynchronize(event3);
			        hipEventSynchronize(event4);
				hipEventElapsedTime(&dt_ms, event3, event4);
			        gpu_t[1] += (float) dt_ms;


				clock_gettime(CLOCK_REALTIME,&timerN);
				//-------------Save energies------------//
				for (i = 0; i < Tinc; ++i)
				{
					pId_loc = perm_search(perm_func, Tinc, i);
					hist_dat[i].energies[k+j*ESs] = Ea[pId_loc];
					hist_dat[i].replica_array[k+j*ESs] = pId_loc;
					hist_dat[i].timestamp_array[k+j*ESs] = (double)((timerN.tv_nsec*0.000000001 + timerN.tv_sec) - (timer0.tv_nsec*0.000000001 + timer0.tv_sec));
                		}

			}
		//Run streaming output
		streaming_output(file1,Ea,Reps_Ex,hist_dat,Tinc,j,ESs,RepSw);
	}

// ----------------------------------------------------------------------------//
	//*************************************************************//
        //**************PRINT HISTOGRAM DATA TO FILES******************//
        //*************************************************************//
	for (i = 0; i < Tinc; ++i)
	{
	hist_dat[i].print_data(dir);
        }
	//*************************************************************//
        //**************CALCULATE FINAL ENERGIES AND END***************//
        //*************************************************************//

       CalcSa_Energy_cuda(Sa,Ea,N,Tinc,blocks,threads,gpu_t);//Calc energy of new states
// ----------------------------------------------------------------------------//

        //************FREE MEMORY***********//

	hist_dat[0].freeEalloc();
        free(perm_func);
	free(Reps_Ex); free(hist_dat);
	hipFree(d_Sa); hipFree(d_Ea); hipFree(d_EaT); 
	cout << "|---------END REMC RUN----------|" << "\n";
	file1 << "\n|------------------REMC FINISHED------------------|\n";
}

//___________________________________________________________________//
// ----------------------------------------------------------------- //
// -----------Calculate Enery Of State(EACH REPLICA)---------------- //
// ----------------------------------------------------------------- //
//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++//
extern void CalcSa_Energy_cuda(int *Sa,double *Ea,int N,int Tinc,int blocks,int threads,float *gpu_t)
{
        int *d_Sa;
        double *d_Ea,*EaT,*d_EaT;
	//hipSetDevice(1);

        //*****MEMORY ALLOCATIONS CPU******
        EaT = (double *)malloc(Tinc * N * N *sizeof(double));

        //*****MEMORY ALLOCATIONS GPU*******    
        hipError_t(hipMalloc ((void **) &d_Sa, Tinc * N * N * sizeof(int)));
        hipError_t(hipMalloc ((void **) &d_Ea, Tinc * sizeof(double)));
        hipError_t(hipMalloc ((void **) &d_EaT, Tinc * N * N * sizeof(double)));
	hipError_t codeMemAlloc = hipGetLastError();
        if (codeMemAlloc != hipSuccess)
                printf("Cuda error ALLOCATIONS -- %s\n",hipGetErrorString(codeMemAlloc));

	//*****Create events for GPU timer*****
        hipEvent_t event5, event6;
        hipEventCreate(&event5);
        hipEventCreate(&event6);

        /* ... Load CPU data into GPU buffers  */
        hipError_t(hipMemcpy(d_Sa,Sa,Tinc * N * N * sizeof(int),hipMemcpyHostToDevice));
        hipError_t codeMemA2 = hipGetLastError();
        if (codeMemA2 != hipSuccess)
                printf("Cuda error INITIAL MEM 1 ALLOCATION COPY -- %s\n",hipGetErrorString(codeMemA2));
        
	hipError_t(hipMemcpy(d_Ea,Ea,Tinc * sizeof(double),hipMemcpyHostToDevice));
        hipError_t codeMemA3 = hipGetLastError();
        if (codeMemA3 != hipSuccess)
                printf("Cuda error INITIAL MEM 2 ALLOCATION COPY -- %s\n",hipGetErrorString(codeMemA3));

	hipError_t(hipMemcpy(d_EaT,EaT,Tinc * N * N * sizeof(double),hipMemcpyHostToDevice));
        hipError_t codeMemA1 = hipGetLastError();
        if (codeMemA1 != hipSuccess)
                printf("Cuda error INITIAL MEM 3 ALLOCATION COPY -- %s\n",hipGetErrorString(codeMemA1));

        //*****RUN PROGRAMS(EVENTS TIME GPU CALCULATIONS)****
        hipEventRecord(event5,0);
        __CalcE_array_cuda__ <<<blocks,threads>>> (d_Sa,d_EaT);
	hipDeviceSynchronize();
        hipError_t code = hipGetLastError();
        if (code != hipSuccess)
                printf("Cuda error -- %s\n",hipGetErrorString(code));


        __sumE_array_cuda__ <<<Tinc,1>>> (d_Ea,d_EaT);
        hipDeviceSynchronize();
	hipError_t code1 = hipGetLastError();
        if (code1 != hipSuccess)
                printf("Cuda error -- %s\n",hipGetErrorString(code1));

        hipEventRecord(event6,0);

        /* ... Transfer data from GPU to CPU */
        hipMemcpy(Ea,d_Ea,Tinc * sizeof(double),hipMemcpyDeviceToHost);

        //Sync the events so that 2 doesnt finish before the end of computations.
        hipEventSynchronize(event5);
        hipEventSynchronize(event6);

        float dt_ms;
        hipEventElapsedTime(&dt_ms, event5, event6);
        gpu_t[2] += (float) dt_ms;

        //************FREE MEMORY***********
        free(EaT);
        hipFree(d_Sa); hipFree(d_Ea); hipFree(d_EaT);
}

